#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <iostream>
#include <fstream>
#include "npy.hpp"
#include "kernels.cu"
#include <chrono>

using namespace std;


void load_data(
    std::string data_dir,
    int n, int w, int h,
    float *depths,
    float *normals,
    float *intr_matrices,
    float *extr_matrices,
    float *intr_matrices_inv,
    float *extr_matrices_inv
) {

    std::vector<unsigned long> shape{};
    bool fortran_order;
    std::vector<float> data;

    auto start_time = std::chrono::steady_clock::now();

    for (int i = 0; i < n; i++) {
        string depth_path = data_dir + "/depths/" + to_string(i + 1) + ".npy";
        npy::LoadArrayFromNumpy(depth_path, shape, fortran_order, data);
        copy(data.data(), data.data() + h * w, depths + i * h * w);

        string normal_path = data_dir + "/normals/" + to_string(i + 1) + ".npy";
        npy::LoadArrayFromNumpy(normal_path, shape, fortran_order, data);
        copy(data.data(), data.data() + h * w * 3, normals + i * h * w * 3);
    }


    string intr_path = data_dir + "/intrinsic.npy";
    npy::LoadArrayFromNumpy(intr_path, shape, fortran_order, data);
    copy(data.data(), data.data() + 9 * n, intr_matrices);

    string extr_path = data_dir + "/extrinsic.npy";
    npy::LoadArrayFromNumpy(extr_path, shape, fortran_order, data);
    copy(data.data(), data.data() + 16 * n, extr_matrices);

    string intr_inv_path = data_dir + "/intrinsic_inv.npy";
    npy::LoadArrayFromNumpy(intr_inv_path, shape, fortran_order, data);
    copy(data.data(), data.data() + 9 * n, intr_matrices_inv);

    string extr_inv_path = data_dir + "/extrinsic_inv.npy";
    npy::LoadArrayFromNumpy(extr_inv_path, shape, fortran_order, data);
    copy(data.data(), data.data() + 16 * n, extr_matrices_inv);


    auto end_time = std::chrono::steady_clock::now();
    
    std::chrono::duration<double> diff = end_time - start_time;
    double seconds = diff.count();

    ofstream myfile;
    string filename = "output/" + std::to_string(n) + "_times.txt";
    myfile.open (filename, std::ios_base::app);
    myfile << "Data Time = " << seconds << " seconds\n";
    myfile.close();
}


__global__ void image2world(
    int n, int w, int h, 
    float *xyzs, float *depths, 
    float *intr_mats_inv, float *extr_mats
) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    
    for (int img = 0; img < n; img++) {
        float depth = depths[img * w * h + i * w + j];
        
        float *xyz = xyzs + 3 * (img * w * h + i * w + j); 
        float *intr_inv = intr_mats_inv + img * 9;
        float *extr = extr_mats + img * 16;

        float3 cam = {
            j * depth * intr_inv[0] + i * depth * intr_inv[1] + depth * intr_inv[2],
            j * depth * intr_inv[3] + i * depth * intr_inv[4] + depth * intr_inv[5],
            j * depth * intr_inv[6] + i * depth * intr_inv[7] + depth * intr_inv[8],
        };

        xyz[0] = extr[0] * cam.x + extr[1] * cam.y + extr[2] * cam.z + extr[3];
        xyz[1] = extr[4] * cam.x + extr[5] * cam.y + extr[6] * cam.z + extr[7];
        xyz[2] = extr[8] * cam.x + extr[9] * cam.y + extr[10] * cam.z + extr[11];
    }
}



__global__ void compute_irradiances(
    int n, int w, int h,
    float *xyzs, float *depths, float *normals,
    float *intr_mats, float *extr_mats,
    float *intr_mats_inv, float *extr_mats_inv,
    float *total_irradiance
) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    for (int cam_i = 0; cam_i < n; cam_i++) {
        for (int cam_j = 0; cam_j < n; cam_j++) {
            // world to camera
            float *xyz_i = xyzs + 3 * (cam_i * w * h + i * w + j);
            float *extr_inv_j = extr_mats_inv + cam_j * 16;
            float *intr_j = intr_mats + cam_j * 9;

            float3 cam = {
                extr_inv_j[0] * xyz_i[0] + extr_inv_j[1] * xyz_i[1] + extr_inv_j[2] * xyz_i[2] + extr_inv_j[3],
                extr_inv_j[4] * xyz_i[0] + extr_inv_j[5] * xyz_i[1] + extr_inv_j[6] * xyz_i[2] + extr_inv_j[7],
                extr_inv_j[8] * xyz_i[0] + extr_inv_j[9] * xyz_i[1] + extr_inv_j[10] * xyz_i[2] + extr_inv_j[11]
            };

            float3 xyd = {
                intr_j[3] * cam.x + intr_j[4] * cam.y + intr_j[5] * cam.z,
                intr_j[0] * cam.x + intr_j[1] * cam.y + intr_j[2] * cam.z,
                intr_j[6] * cam.x + intr_j[7] * cam.y + intr_j[8] * cam.z
            };

            int x_j = (int) (xyd.x / xyd.z) ;
            int y_j = (int) (xyd.y / xyd.z);
            float d_j = xyd.z;
            float depth_j = -1.;

            if (x_j >= 0 && x_j < h && y_j >=0 && y_j < w) {
                depth_j = depths[cam_j * w * h + x_j * w + y_j];
            }

            float *normal_i = normals + 3 * (cam_i * w * h + i * w + j);
            float3 displacement_other = {
                xyz_i[0] - extr_mats[cam_j * 16 + 3],
                xyz_i[1] - extr_mats[cam_j * 16 + 7],
                xyz_i[2] - extr_mats[cam_j * 16 + 11]
            };

            float direction_norm_sqr = 
                displacement_other.x * displacement_other.x + 
                displacement_other.y * displacement_other.y + 
                displacement_other.z * displacement_other.z;
            
            float direction_norm = sqrt(direction_norm_sqr);

            float3 neg_direction_other = {
                displacement_other.x / -direction_norm,
                displacement_other.y / -direction_norm,
                displacement_other.z / -direction_norm,
            };

            float cosine_law = 
                neg_direction_other.x * normal_i[0] + 
                neg_direction_other.y * normal_i[1] +
                neg_direction_other.z * normal_i[2];

            if (depth_j > d_j-0.01 && d_j > 0. && cosine_law > 0)
                total_irradiance[cam_i * w * h + i * w + j] += cosine_law * (1. / direction_norm_sqr);
        }
    }
}

int main(int argc, char **argv)
{
    printf("Running %d frames...\n", stoi(argv[1]));

    // You can directly tune n for profiling
    // but h and w aren't changable unless 
    // you make changes to the data as well
    // int n = 100;
    int n = stoi(argv[1]);
    int h = 1024;
    int w = 1024;

    float *depths_cpu = (float *) malloc(sizeof(float) * n * w * h);
    float *normals_cpu = (float *) malloc(sizeof(float) * n * w * h * 3);
    float *xyzs_cpu = (float *) malloc(sizeof(float) * n * w * h * 3);
    float *intr_matrices_cpu = (float *) malloc(sizeof(float) * n * 9);
    float *extr_matrices_cpu = (float *) malloc(sizeof(float) * n * 16);
    float *intr_matrices_inv_cpu = (float *) malloc(sizeof(float) * n * 9);
    float *extr_matrices_inv_cpu = (float *) malloc(sizeof(float) * n * 16);
    float *total_irradiance_cpu = (float *) malloc(sizeof(float) * n * w * h);

    string path{"/home/haodali/cs267_final_project/dataset_100"};
    load_data(path, n, w, h, 
        depths_cpu, normals_cpu, 
        intr_matrices_cpu, extr_matrices_cpu, 
        intr_matrices_inv_cpu, extr_matrices_inv_cpu
    );
    

    float *depths;
    hipMalloc((void**) &depths, sizeof(float) * n * w * h);
    hipMemcpy(depths, depths_cpu, sizeof(float) * n * w * h, hipMemcpyHostToDevice);
    
    float *normals;
    hipMalloc((void**) &normals, sizeof(float) * n * w * h * 3);
    hipMemcpy(normals, normals_cpu, sizeof(float) * n * w * h * 3, hipMemcpyHostToDevice);

    float *xyzs;
    hipMalloc((void**) &xyzs, sizeof(float) * n * w * h * 3);
    
    float *intr_matrices;
    hipMalloc((void**) &intr_matrices, sizeof(float) * n * 9);
    hipMemcpy(intr_matrices, intr_matrices_cpu, sizeof(float) * n * 9, hipMemcpyHostToDevice);
    
    float *extr_matrices;
    hipMalloc((void**) &extr_matrices, sizeof(float) * n * 16);
    hipMemcpy(extr_matrices, extr_matrices_cpu, sizeof(float) * n * 16, hipMemcpyHostToDevice);
    
    float *intr_matrices_inv;
    hipMalloc((void**) &intr_matrices_inv, sizeof(float) * n * 9);
    hipMemcpy(intr_matrices_inv, intr_matrices_inv_cpu, sizeof(float) * n * 9, hipMemcpyHostToDevice);
    
    float *extr_matrices_inv; 
    hipMalloc((void**) &extr_matrices_inv, sizeof(float) * n * 16);
    hipMemcpy(extr_matrices_inv, extr_matrices_inv_cpu, sizeof(float) * n * 16, hipMemcpyHostToDevice);

    float *total_irradiance;
    hipMalloc((void **)&total_irradiance, sizeof(float) * n * w * h);
    hipMemset(total_irradiance, 0., n * w * h);

    dim3 n_blocks(32, 32);
    dim3 n_threads(32, 32);
    
    auto start_time = std::chrono::steady_clock::now();
    image2world<<<n_blocks, n_threads>>>(n, w, h, xyzs, depths, intr_matrices_inv, extr_matrices);
    hipDeviceSynchronize();

    compute_irradiances<<<n_blocks, n_threads>>>(
        n, w, h,
        xyzs, depths, normals, 
        intr_matrices, extr_matrices,
        intr_matrices_inv, extr_matrices_inv,
        total_irradiance
    );
    hipDeviceSynchronize();

    auto end_time = std::chrono::steady_clock::now();

    std::chrono::duration<double> diff = end_time - start_time;
    double seconds = diff.count();

    ofstream myfile;
    string filename = "output/" + std::to_string(n) + "_times.txt";
    myfile.open (filename, std::ios_base::app);
    myfile << "Simulation Time = " << seconds << " seconds\n";
    myfile.close();

    hipMemcpy(total_irradiance_cpu, total_irradiance, sizeof(float) * n * w * h, hipMemcpyDeviceToHost);
    hipMemcpy(xyzs_cpu, xyzs, sizeof(float) * n * w * h * 3, hipMemcpyDeviceToHost);

    const std::vector<size_t> shape{n, h, w};
    npy::SaveArrayAsNumpy("irr.npy", false, shape.size(), shape.data(), total_irradiance_cpu);
    
}
